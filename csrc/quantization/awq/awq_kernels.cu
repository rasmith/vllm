#include "hip/hip_runtime.h"
#include <torch/all.h>
#include <c10/cuda/CUDAGuard.h>

#include <hip/hip_fp16.h>

#define CDIV(A, B) (((A) + (B)-1) / (B))

__device__ void AtomicAdd(half* address, half val) {
  uint32_t* address_as_ui =
      (uint32_t*)((size_t)address - ((size_t)address & 2));
  uint32_t old = *address_as_ui;
  uint32_t assumed;

  do {
    assumed = old;
    __half_raw hsum;
    hsum.x = (size_t)address & 2 ? (old >> 16) : (old & 0xffff);
    half temp = __hadd(hsum, val);
    hsum = __half_raw(temp);
    old = (size_t)address & 2 ? (old & 0xffff) | (hsum.x << 16)
                              : (old & 0xffff0000) | hsum.x;
    old = atomicCAS(address_as_ui, assumed, old);
  } while (assumed != old);
}

using float16_t = _Float16;
using float32_t = float;

using float16x4_t = float16_t __attribute__((ext_vector_type(4)));
using float32x4_t = float32_t __attribute__((ext_vector_type(4)));

template <int N>
struct HalfN {
  half h[N];
};

template <int TILE_WIDTH>
__global__ __launch_bounds__(128) void awq_gemm_mfma_kernel(
    half* a, int* q, int* zeros, half* scales, int size_n, int size_k,
    int size_m, int group_size, int split_k, half* c) {
  __constant__ static const int kReverseAwqLookup[8] = {0, 4, 1, 5, 2, 6, 3, 7};

  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int k = blockIdx.z;  // Get k for splitK

  // dim3 = (16, 4, splitK) = 64 * splitK threads per block, splitK waves.

  // NOTE: Even though the block size is (16, 4, splitK), each (16,4) group
  // is responsible for a 16 x 16 tile and has the 16 * 4 threads necessary to
  // do the work (mfma). So, below uses TILE_WIDTH and TILE_WIDTH needs to be
  // 16, at least for now.

  // NOTE: The outputs will be (row, col), (row + 1, col), (row + 2, col)
  // and (row + 3, col).

  // The row is the row to work on for pulling values for mfma from.
  int row = blockIdx.x * TILE_WIDTH + tx;
  // The output column that will be used and also used for pulling mfma values
  // from the quantized matrix.
  int col = blockIdx.y * TILE_WIDTH + tx;

  int num_tiles = CDIV(size_k, TILE_WIDTH);

  __shared__ int z_values[TILE_WIDTH / 8];  // These are zero values.
  __shared__ half s_values[TILE_WIDTH];     // These are scale values.
  __shared__ int q_values[TILE_WIDTH]
                         [TILE_WIDTH / 8];  // These are quantized AWQ values.

  // Vectorized values, correspond to registers, cannot take address of any of
  // these.  I tried, and the compiler told me no.
  float16x4_t a_frag{0.0, 0.0, 0.0, 0.0};
  float16x4_t b_frag(0.0);
  float32x4_t accumulator(0.0);

  //  Have CDIV(size_k, split_k * TILE_WIDTH) groups of tiles:
  //
  //   --------------------------------------------
  //        group_0      |       group_1     | ...
  //   --------------------------------------------
  //   0 ... split_k - 1 | 0 ... split_k - 1 | ...
  //   --------------------------------------------
  //
  //  and this thread will work on tiles with "tile index"
  //     threadIdx.z + i * split_k
  // for i = 0, .. , cdiv(size_k, split_k * TILE_WIDTH)
  //
  for (int tile = k; tile < num_tiles; tile += split_k) {
    if (row >= 0 && row < size_n) {
      int base_col = tile * TILE_WIDTH + 4 * ty;
      int offset = row * size_k + base_col;

      // Load 8 bytes at a time for better memory performance, if possible.
      // Need to deal with the case where the tile falls off the
      // edge though, but still trying to load contiguous values whenever
      // possible, with the hope that some coalescing will occur along with
      // vectorized load.

      // NOTE: Could try using uint2 maybe or some other vectorized type.
      // Could also try loading into shared memory first.
      if (base_col < size_k) {
        HalfN<4> h = *reinterpret_cast<HalfN<4>*>(&a[offset]);
        a_frag[0] = h.h[0];
        a_frag[1] = h.h[1];
        a_frag[2] = h.h[2];
        a_frag[3] = h.h[3];
      } else if (base_col + 2 < size_k) {
        HalfN<3> h = *reinterpret_cast<HalfN<3>*>(&a[offset]);
        a_frag[0] = h.h[0];
        a_frag[1] = h.h[1];
        a_frag[2] = h.h[2];
        a_frag[3] = __ushort_as_half(0);
      } else if (base_col + 1 < size_k) {
        HalfN<2> h = *reinterpret_cast<HalfN<2>*>(&a[offset]);
        a_frag[0] = h.h[0];
        a_frag[1] = h.h[1];
        a_frag[2] = __ushort_as_half(0);
        a_frag[3] = __ushort_as_half(0);
      } else if (base_col < size_k) {
        a_frag[0] = a[offset];
        a_frag[1] = __ushort_as_half(0);
        a_frag[2] = __ushort_as_half(0);
        a_frag[3] = __ushort_as_half(0);
      }
    } else {
      a_frag[0] = __ushort_as_half(0);
      a_frag[1] = __ushort_as_half(0);
      a_frag[2] = __ushort_as_half(0);
      a_frag[3] = __ushort_as_half(0);
    }

    // This was the code for above, but using the above gave a nice improvement.
    // Keeping around for now, since below is more readable, with "readibility"
    // benig a relative concept in this scenario.

    //#pragma unroll
    // for (int i = 0; i < 4; ++i) {
    //// Go down to the current row, and then over to the current k-tile
    //// and then get the 4 values starting at:
    ////    (row, tile * TILE_WIDTH + 4 * ty)
    //// and load them into registers.
    // int a_j = tile * TILE_WIDTH + 4 * ty + i;
    // int a_i = row;

    // if (a_i >= 0 && a_i < size_n && a_j >= 0 && a_j < size_k) {
    // a_frag[i] = a[a_i * size_k + a_j];
    //} else {
    // a_frag[i] = __ushort_as_half(0);
    //}
    //}

    // OK, threads per block are (x, y, z) = (16, 4, splitK).
    // Have y = 0  , x = 0:15, z = 0:splitK - 1 load the zeros.
    // Have y = 1  , x = 0:15, z = 0:splitK - 1 load the scales.
    // Have y = 2:3, x = 0:15, z = 0:splitK - 1 load quantized values from q.
    // Hopefully, this hides some latency and helps minimize divergence.
    //
    // NOTE: This gets loaded into shared memory, and zeros are recorded
    // when out of bounds access would have happened, so the mfma code
    // can just load straight values.
    //
    // NOTE: Where the quantized values below are being loaded, could load
    // those into registers, and then do a cross-lane exchange after the
    // if-statement.  This would eliminate the need for the q_values array.
    // Might need an additional synchthreads.
    if (ty == 0) {
      int z_row = tile * TILE_WIDTH / group_size;
      int z_col = col / 8;
      if (z_row < size_k / group_size && z_col < size_m / 8) {
        z_values[tx / 8] = zeros[z_row * (size_m / 8) + z_col];
      } else {
        z_values[tx / 8] = 0;
      }
    } else if (ty == 1) {
      int s_row = tile * TILE_WIDTH / group_size;
      int s_col = col;
      if (s_row < size_k / group_size && s_col < size_m) {
        s_values[tx] = scales[s_row * size_m + s_col];
      } else {
        s_values[tx] = __ushort_as_half(0);
      }
    } else {
      int q_row = tile * TILE_WIDTH + tx;
      int q_col = blockIdx.y * TILE_WIDTH / 8 + ty % 2;
      if (q_row < size_k && q_col < size_m / 8) {
        q_values[tx][ty % 2] = q[q_row * (size_m / 8) + q_col];
      } else {
        q_values[tx][ty % 2] = 0;
      }
    }

    __syncthreads();

// if (blockIdx.x == 0 && blockIdx.y == 0) {
// printf("(%d, %d) -> a_fragment: [%.3f, %.3f, %.3f, %.3f]\n", threadIdx.x,
// threadIdx.y, __half2float(a_frag[0]), __half2float(a_frag[1]),
//__half2float(a_frag[2]), __half2float(a_frag[3]));
//}
#pragma unroll
    for (int i = 0; i < 4; ++i) {
      // Go down to the current k-tile and then get the 4 values starting at:
      //    (tile * TILE_WIDTH + 4 * ty, col)
      // and load them into registers.

      // Although, really we're just getting the values from LDS.

      int b_j = col;
      int b_i = tile * TILE_WIDTH + 4 * ty + i;

      // NOTE: Loading values in this way seems not great, would like to load
      // contiguous values.  Could load contiguously and then use cross-lane
      // intrinsics to exchange values. Basically, load directly into registers,
      // and then shuffle or DPP the values into the threads that need them.
      //
      // Could also just try shared memory, but I think cross-lane is more
      // efficient since moving the data into registers and then exchanging
      // will have less latency.
      int q_value = q_values[4 * ty + i][tx / 8];

      int z_value = z_values[tx / 8];
      int shift = kReverseAwqLookup[b_j % 8] * 4;
      int b_int4 = (q_value >> shift) & 0xF;
      int z_int4 = (z_value >> shift) & 0xF;
      half scale = s_values[tx];

      // Since AWQ quantized, actually need to dequantize first.
      b_frag[i] = __int2half_rn(b_int4 - z_int4) * scale;
    }
    // if (blockIdx.x == 0 && blockIdx.y == 0) {
    // printf("[%d] (%d, %d) -> b_fragment: [%.3f, %.3f, %.3f, %.3f]\n", col,
    // threadIdx.x, threadIdx.y, __half2float(b_frag[0]),
    //__half2float(b_frag[1]), __half2float(b_frag[2]),
    //__half2float(b_frag[3]));
    //}

    accumulator = __builtin_amdgcn_mfma_f32_16x16x16f16(a_frag, b_frag,
                                                        accumulator, 0, 0, 0);
  }

// printf("[%d, %d] %.3f %.3f %.3f %.3f\n", tx, ty,
// __half2float(accumulator[0]),
//__half2float(accumulator[1]), __half2float(accumulator[2]),
//__half2float(accumulator[3]));
#pragma unroll
  for (int i = 0; i < 4; ++i) {
    // Starting row is (row / TILE_WIDTH) * TILE_WIDTH for this c-tile.
    // So go to ((row / TILE_WIDTH) * TILE_WIDTH + 4 * ty, blockIdx.y *
    // TILE_WIDTH + tx) and start writing values down the column.
    int c_j = col;
    int c_i = (row / TILE_WIDTH) * TILE_WIDTH + ty * 4 + i;

    // NOTE: Same idea as with loading the quantized values, this memory
    // access pattern seems bad, so could try using cross-lane, and then
    // storing contiguous values.
    if (c_i >= 0 && c_i < size_n && c_j >= 0 && c_j < size_m) {
      c[k * size_m * size_n + c_i * size_m + c_j] = accumulator[i];
    }
  }
  __syncthreads();
}

template <int TILE_WIDTH>
__global__ void awq_gemm_kernel(half* a, int* q, int* zeros, half* scales,
                                int size_n, int size_k, int size_m,
                                int group_size, int split_k, half* c) {
  static const int kReverseAwqLookup[8] = {0, 4, 1, 5, 2, 6, 3, 7};
  float output = 0.0f;
  int row = blockIdx.y * TILE_WIDTH + threadIdx.y;
  int col = blockIdx.x * TILE_WIDTH + threadIdx.x;
  int k = blockIdx.z;

  __shared__ half a_tile[TILE_WIDTH][TILE_WIDTH];
  __shared__ half b_tile[TILE_WIDTH][TILE_WIDTH];

  int tile_start = 0;
  int tile_end = CDIV(size_k, TILE_WIDTH);
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int ay = row;
  int bx = col;
  int num_tiles = CDIV(size_k, TILE_WIDTH);

  for (int tile = k; tile < num_tiles; tile += split_k) {
    int ax = tile * TILE_WIDTH + tx;
    if (ay < size_n && ax < size_k) {
      a_tile[ty][tx] = a[ay * size_k + ax];
    } else {
      a_tile[ty][tx] = __ushort_as_half(0);
    }

    int by = tile * TILE_WIDTH + ty;

    if (by < size_k && bx < size_m) {
      half scale = scales[(by / group_size) * size_m + bx];
      int q_value = q[(by / group_size) * (size_m / 8) + (bx / 8)];
      int z_value = zeros[(by / group_size) * (size_m / 8) + (bx / 8)];
      int shift = kReverseAwqLookup[bx % 8] * 4;
      int b_int4 = (q_value >> shift) & 0xF;
      int z_int4 = (z_value >> shift) & 0xF;
      b_tile[ty][tx] = __int2half_rn(b_int4 - z_int4) * scale;
    } else {
      b_tile[ty][tx] = __ushort_as_half(0);
    }

    __syncthreads();

    for (int k = 0; k < TILE_WIDTH; ++k) {
      output += __half2float(a_tile[ty][k]) * __half2float(b_tile[k][tx]);
    }
    __syncthreads();
  }

  if (row < size_n && col < size_m) {
    c[k * size_n * size_m + row * size_m + col] = __float2half_rn(output);
  }
}

// in_feats: M, IC [float16]
// kernel: IC, OC // 8 [int32] -> cast to IC, OC [uint4b]
// scaling_factors: IC // G, OC [float16]
// zeros: IC // G, OC // 8 [int32] -> cast to IC // G, OC [uint4b]
// assume that batch_size < 16 for now

torch::Tensor awq_gemm_test(torch::Tensor input_tensor,
                            torch::Tensor qweight_tensor,
                            torch::Tensor scales_tensor,
                            torch::Tensor qzeros_tensor, int64_t splitK) {
  int size_n = input_tensor.size(0);
  int size_k = qweight_tensor.size(0);
  int size_m = qweight_tensor.size(1) * 8;
  int group_size = qweight_tensor.size(0) / qzeros_tensor.size(0);
  const at::cuda::OptionalCUDAGuard device_guard(device_of(input_tensor));

  auto options = torch::TensorOptions()
                     .dtype(input_tensor.dtype())
                     .device(input_tensor.device());
  at::Tensor result_tensor = torch::zeros({splitK, size_n, size_m}, options);

  half* input = reinterpret_cast<half*>(input_tensor.data_ptr<at::Half>());
  int* qweights = reinterpret_cast<int*>(qweight_tensor.data_ptr<int>());
  half* scales = reinterpret_cast<half*>(scales_tensor.data_ptr<at::Half>());
  int* qzeros = reinterpret_cast<int*>(qzeros_tensor.data_ptr<int>());
  half* c = reinterpret_cast<half*>(result_tensor.data_ptr<at::Half>());

  const hipStream_t stream = at::cuda::getCurrentCUDAStream();

  constexpr bool kUseMfma = true;
  constexpr int kTileWidth = (kUseMfma ? 16 : 32);
  if (kUseMfma) {
    dim3 threads_per_block(16, 4);
    dim3 blocks(CDIV(size_n, kTileWidth), CDIV(size_m, kTileWidth),
                splitK);  // CDIV(size_m, kTileWidth));
    // std::cout << "threads_per_block.x = " << threads_per_block.x
    //<< ", threads_per_block.y = " << threads_per_block.y << "\n";
    // std::cout << "blocks.x = " << blocks.x
    //<< ", blocks.y = " << blocks.y << "\n";

    awq_gemm_mfma_kernel<kTileWidth><<<blocks, threads_per_block, 0, stream>>>(
        input, qweights, qzeros, scales, size_n, size_k, size_m, group_size,
        splitK, c);
  } else {
    dim3 threads_per_block(kTileWidth, kTileWidth);
    dim3 blocks(CDIV(size_m, kTileWidth), CDIV(size_n, kTileWidth),
                splitK);  // CDIV(size_m, kTileWidth));
    // std::cout << "threads_per_block.x = " << threads_per_block.x
    //<< ", threads_per_block.y = " << threads_per_block.y
    //<< ", threads_per_block.z = " << threads_per_block.z << "\n";
    // std::cout << "blocks.x = " << blocks.x << ", blocks.y = " << blocks.y
    //<< "\n";
    // std::cout << "Launching kernel...\n";

    awq_gemm_kernel<kTileWidth><<<blocks, threads_per_block, 0, stream>>>(
        input, qweights, qzeros, scales, size_n, size_k, size_m, group_size,
        splitK, c);
  }
  return result_tensor.sum(0);
}
